#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>

char **negative;
char **positive;
char **test;
char **stop;
int posLength;
int negLength;
int testLength;
int stopLength;


int negation(char *key) {
	char temp[10] = "no";
	if (strcmp(key, temp) == 0)
		return 1;
	char temp2[10] = "not";
	if (strcmp(key, temp2) == 0)
		return 1;
	return 0;
}


int checkStopwords(char *key) {
	int i;
	int temp = 0;
	for (i = 0; i<stopLength; i++) {
		if (strcmp(stop[i], key) == 0)
			temp += 1;
	}
	return temp;
}

int findCountPos(char *key) {
	int i, count = 0;
	for (i = 0; i<posLength; i++) {
		//printf("%d %s %s\n", strcmp(key, positive[i]) == 0,key,positive[i]);
		if (strcmp(key, positive[i]) == 0)
			count=count+1;
	}
	printf("%s %d\n", key, count);
	return count;
}

int findCountNeg(char *key) {
	int i, count = 0;
	for (i = 0; i<negLength; i++) {
		if (strcmp(key, negative[i]) == 0)
			count=count+1;
	}
	printf("%s %d\n", key, count);
	return count;
}


void readStop() {
	FILE *fp;
	int count = 0;
	char str[10];

	int i, n, stringsize;
	char a[100];

	//char **temp=negative;
	fp = fopen("G:\\test\\pc\\pc-cuda\\pc-cuda\\files\\stopWords.txt", "r");

	while (fscanf(fp, "%s", str) != EOF) {
		count = count + 1;
		if (count >10000)
			break;
	}
	stop = (char **)malloc(count * sizeof(char *));
	fclose(fp);
	fp = fopen("G:\\test\\pc\\pc-cuda\\pc-cuda\\files\\stopWords.txt", "r");
	for (i = 0; i < count; i++)
	{
		fscanf(fp, "%s", a);
		stringsize = strlen(a);
		stop[i] = (char *)malloc(stringsize + 1);
		strcpy(stop[i], a);
	}
	fclose(fp);
	stopLength = count;
	//print(3);
}


void readTest() {
	int count = 0, i, stringsize;
	char temp[1000], temp2[1000];
	char a[100];
	printf("Enter Sentence to be processed :");
	fgets(temp, sizeof(temp), stdin);
	//printf("%s \n",temp);
	strcpy(temp2, temp);
	char *token = strtok(temp, " ");
	while (token != NULL) {
		count++;
		//printf("%s\n",token );
		//strcpy(a,token);
		//printf("%s\n",a );
		/*
		stringsize=strlen(a);
		test[i] = (char *)malloc(stringsize+2);
		strcpy(test[i],a);
		*/
		token = strtok(NULL, " ");
	}
	//printf("%d\n",count );
	test = (char **)malloc(count * sizeof(char *));
	strcpy(temp, temp2);
	//printf("%s\n",temp );
	token = strtok(temp, " ");
	for (i = 0; i<count; i++) {
		strcpy(a, token);
		//printf("%s\n",a );
		stringsize = strlen(a);
		test[i] = (char *)malloc(stringsize );
		strcpy(test[i], a);
		token = strtok(NULL, " ");
		//printf("Hello World\n%s",token );
	}
	//printf("%d\n",count );

	//token=(char *)strtok(temp," ");
	testLength = count;
	//print(2);
}

void readpos() {
	FILE *fp;
	int count = 0;
	char str[10];

	int i, n, stringsize;
	char a[100];

	//char **temp=negative;
	fp = fopen("G:\\test\\pc\\pc-cuda\\pc-cuda\\files\\positive.txt", "r");

	while (fscanf(fp, "%s", str) != EOF) {
		count = count + 1;
		if (count >10000)
			break;
	}
	positive = (char **)malloc(count * sizeof(char *));
	fclose(fp);
	fp = fopen("G:\\test\\pc\\pc-cuda\\pc-cuda\\files\\positive.txt", "r");
	for (i = 0; i < count; i++)
	{
		fscanf(fp, "%s", a);
		stringsize = strlen(a);
		positive[i] = (char *)malloc(stringsize + 1);
		strcpy(positive[i], a);
	}
	fclose(fp);
	//printf("Hello World \n %d\n",count );
	posLength = count;
	//print(1);
}

void readneg() {
	FILE *fp;
	int count = 0;
	char str[10];

	int i, n, stringsize;
	char a[100];

	//char **temp=negative;
	fp = fopen("G:\\test\\pc\\pc-cuda\\pc-cuda\\files\\negative.txt", "r");

	while (fscanf(fp, "%s", str) != EOF) {
		count = count + 1;
		if (count >10000)
			break;
	}
	negative = (char **)malloc(count * sizeof(char *));
	fclose(fp);
	fp = fopen("G:\\test\\pc\\pc-cuda\\pc-cuda\\files\\negative.txt", "r");
	for (i = 0; i < count; i++)
	{
		fscanf(fp, "%s", a);
		stringsize = strlen(a);
		negative[i] = (char *)malloc(stringsize + 1);
		strcpy(negative[i], a);
	}
	fclose(fp);
	//printf("Hello World \n %d\n",count );
	negLength = count;
	//print(0);
}

__global__ void findcount(char **negative, char *key,int *count , int n) {
	int i = threadIdx.x;
	if (i < n)
	{
		printf("%s and %s in %d and %d\n",negative[i][0],key, i,count[0]);
		int j = 0,flag=0;
		while (key[j] != '\0') {
			if (key[j] != negative[i][j]) {
				flag = 1;
				break;
			}
		}
		if (flag == 1) {
			count[0] = count[0] + 1;
		}
	}
}


__global__ void addarray(float *a, float *b, int n) {
	int i = threadIdx.x;
	if (i < n) {
		b[0] = b[0] + a[i];
		printf(" value in count : %lf thread no. %d and value %d\n", b[0] ,i , a[i]);
	}

}


__global__ void check(float *a, float *b, int *c, float n) {
	int i = threadIdx.x;
	if (i < n) {
		printf(" condition is %d value are %lf , %lf\n", a[i] < b[i],a[i],b[i]);
		if ((a[i] < b[i]) == 0) {
			c[i] = -1;
			printf(" c[%d]values is %d\n", i, c[i]);
		}
		else
		{
			c[i] = 1;
		}
	}

}


int main() {


	
	int posTotal, negTotal, i,j=0, flag = 0;
	float posProb = 0, negProb = 0, posCons, negCons, TotalPos = 0, TotalNeg = 0;
	double pos, neg, k;
	int posarrayCount, negarrayCount;
	float *posarray,*posarray_d;
	float *negarray,*negarray_d;
	int *count;
	int sum=0;
	clock_t t;
	printf("Hello world \n");
	t = clock();
	readneg();
	readpos();
	readTest();
	readStop();


	posTotal = posLength;
	negTotal = negLength;
	posCons = (posTotal*1.0) / (posTotal + negTotal);
	negCons = (negTotal*1.0) / (posTotal + negTotal);

	hipMallocManaged(&posarray, sizeof(float) * testLength);
	hipMallocManaged(&negarray, sizeof(float) * testLength);

	



	for (i = 0; i<testLength; i++) {
		if (negation(test[i]) == 1) {
			flag = 1;
			continue;
		}
		//printf("%s %d\n",test[i] ,checkStopwords(test[i]) );
		if (checkStopwords(test[i]) != 0)
			continue;
		if (flag == 1)
		{
			k = findCountNeg(test[i]);
			pos = log((k + 1) / (negTotal*1.0));
			flag = 0;
		}
		else
		{
			k = findCountPos(test[i]);
			pos = log((k + 1) / (posTotal*1.0));
		}

		//printf("%lf %lf\n",pos,neg);
		posarray[j++] = pos;
		TotalPos += pos;

	}
	posarrayCount = j;
	j = 0;
	for (i = 0; i<testLength; i++) {
		if (negation(test[i]) == 1) {
			flag = 1;
			continue;
		}
		if (checkStopwords(test[i]) != 0)
			continue;
		if (flag == 1)
		{
			k = findCountPos(test[i]);
			neg = log((k + 1) / (posTotal*1.0));
			flag = 0;
		}
		else
		{
			k = findCountNeg(test[i]);
			neg = log((k + 1) / (negTotal*1.0));
		}
		negarray[j++] = neg;
		TotalNeg += neg;
	}
	negarrayCount = j;
	for (i = 0; i < posarrayCount;i++) {
		printf("positive%lf,negative%lf\n",posarray[i],negarray[i]);
	}
	hipMallocManaged(&count, sizeof(int) * posarrayCount);
	check << <1,posarrayCount>> > (posarray,negarray, count,posarrayCount);
	hipDeviceSynchronize();
	for (i = 0; i < posarrayCount; i++) {
		sum += count[i];
		printf("sum is :%d\n",sum);
	}
	if (sum > 0)
		printf("positive\n");
	else
		printf("negative\n");
	
	printf("Total positve %lf\nTotal negative %lf\n",TotalPos,TotalNeg);
}
